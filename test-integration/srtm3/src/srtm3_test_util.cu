#include "hip/hip_runtime.h"
/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */
#include "srtm3_test_util.cuh"

#include "srtm3_elevation_calc.cuh"

namespace alus {
namespace tests {

__global__ void SRTM3AltitudeTester(double* lats, double* lons, double* results, SRTM3TestData data) {
    const int idx = threadIdx.x + (blockDim.x * blockIdx.x);

    if (idx < data.size) {
        results[idx] = snapengine::srtm3elevationmodel::GetElevation(lats[idx], lons[idx], &data.tiles);
    }
}

hipError_t LaunchSRTM3AltitudeTester(dim3 grid_size, dim3 block_size, double* lats, double* lons, double* results,
                                      SRTM3TestData data) {
    SRTM3AltitudeTester<<<grid_size, block_size>>>(lats, lons, results, data);
    return hipGetLastError();
}

}  // namespace tests
}  // namespace alus
