#include "hip/hip_runtime.h"
/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */

#include "calc_kernels.cuh"

#include <cmath>

#include "calc_funcs.h"

namespace alus::math::calckernels {

__global__ void CalcDb(cuda::KernelArray<float> buffer, size_t w, size_t h, float no_data_value) {
    const auto thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const auto thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (thread_x >= w || thread_y >= h) {
        return;
    }

    const auto index = thread_y * w + thread_x;
    const auto orig_value = buffer.array[index];
    buffer.array[index] = calcfuncs::Decibel(orig_value, no_data_value);
}

__global__ void CalcDiv(cuda::KernelArray<float> dividend, cuda::KernelArray<float> divisor, size_t w, size_t h,
                        cuda::KernelArray<float> result, float no_data_value) {
    const auto thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const auto thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (thread_x >= w || thread_y >= h) {
        return;
    }

    const auto index = thread_y * w + thread_x;
    const auto dividend_orig_value = dividend.array[index];
    const auto divisor_orig_value = divisor.array[index];
    if (isnan(dividend_orig_value) || isnan(divisor_orig_value)) {
        result.array[index] = no_data_value;
        return;
    }
    if (!isnan(no_data_value) && (dividend_orig_value == no_data_value || divisor_orig_value == no_data_value)) {
        result.array[index] = no_data_value;
        return;
    }

    if (divisor_orig_value == 0) {
        result.array[index] = 0;
    } else {
        result.array[index] = dividend_orig_value / divisor_orig_value;
    }
}

}  // namespace alus::math::calckernels