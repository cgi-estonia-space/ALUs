/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */

#include "cuda_device_init.h"

#include <stdexcept>
#include <thread>

#include <hip/hip_runtime.h>

#include "cuda_util.h"

namespace alus::cuda {
CudaInit::CudaInit() {
    init_future_ = std::async(std::launch::async, [this]() { this->QueryDevices(); });
}

bool CudaInit::IsFinished() const {
    if (!init_future_.valid()) {
        throw std::runtime_error("The future is already a past, invalid state queried.");
    }
    return init_future_.wait_for(std::chrono::milliseconds(0)) == std::future_status::ready;
}

void CudaInit::QueryDevices() {
    int device_count{};
    CHECK_CUDA_ERR(hipGetDeviceCount(&device_count));
    if (!device_count) {
        throw std::runtime_error("No GPU devices detected");
    }
    for (int i{}; i < device_count; i++) {
        hipDeviceProp_t deviceProp;
        CHECK_CUDA_ERR(hipGetDeviceProperties(&deviceProp, i));
        devices_.emplace_back(i, &deviceProp);
        // Whatever will first start invoking GPU, might be delayed if this thread does not finish.
        // But when waiting, a first invocation of GPU could be delayed by waiting here.
        // Also no error checking is done, because if there are errors, then sooner or later they will pop out
        // somewhere else.
        device_warmups_.emplace_back([i]() {
            hipSetDevice(i);
            hipFree(nullptr);
        });
    }
}

void CudaInit::CheckErrors() {
    if (!init_future_.valid()) {
        throw std::runtime_error("The future is already a past, invalid state queried.");
    }
    init_future_.get();
}

CudaInit::~CudaInit() {
    // Just in case wait if any left hanging.
    if (init_future_.valid()) {
        init_future_.wait_for(std::chrono::seconds(10));
    }

    for (auto& t : device_warmups_) {
        if (t.joinable()) {
            t.join();
        }
    }
}

}  // namespace alus::cuda