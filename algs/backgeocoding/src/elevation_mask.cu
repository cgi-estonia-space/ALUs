#include "hip/hip_runtime.h"
/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */

#include "backgeocoding_constants.h"
#include "cuda_util.h"
#include "dem_calc.cuh"
#include "dem_property.h"
#include "elevation_mask_computation.h"

namespace alus {
namespace backgeocoding {

// regardless whether you mask or not, you need to run the if(data.device_x_points[idx] ... to purify the indexes.
__global__ void ElevationMask(ElevationMaskData data) {
    const size_t idx = threadIdx.x + (blockDim.x * blockIdx.x);

    if (idx < data.size) {
        if (data.device_x_points[idx] == INVALID_INDEX || data.device_y_points[idx] == INVALID_INDEX) {
            data.device_x_points[idx] = INVALID_INDEX;
            data.device_y_points[idx] = INVALID_INDEX;
            return;
        }

        const double lat = data.device_lat_array[idx];
        const double lon = data.device_lon_array[idx];
        const double alt = snapengine::dem::GetElevation(lat, lon, &data.tiles, data.dem_property);

        // TODO: this may need to change if we decide not to use mask.
        if (data.mask_out_area_without_elevation && alt == snapengine::srtm3elevationmodel::NO_DATA_VALUE) {
            data.device_x_points[idx] = INVALID_INDEX;
            data.device_y_points[idx] = INVALID_INDEX;
        } else {
            (*data.not_null_counter)++;  // race condition is not important here.
        }
    }
}

hipError_t LaunchElevationMask(ElevationMaskData data) {
    dim3 block_size(416);
    dim3 grid_size(cuda::GetGridDim(block_size.x, data.size));

    ElevationMask<<<grid_size, block_size>>>(data);
    return hipGetLastError();
}

}  // namespace backgeocoding
}  // namespace alus
