#include "hip/hip_runtime.h"
/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */

#include "backgeocoding_constants.h"
#include "copdem_cog_30m_calc.cuh"
#include "cuda_util.h"
#include "dem_calc.cuh"
#include "dem_property.h"
#include "dem_type.h"
#include "elevation_mask_computation.h"

namespace alus {
namespace backgeocoding {

// regardless whether you mask or not, you need to run the if(data.device_x_points[idx] ... to purify the indexes.
__global__ void ElevationMask(ElevationMaskData data) {
    const size_t idx = threadIdx.x + (blockDim.x * blockIdx.x);

    if (idx < data.size) {
        if (data.device_x_points[idx] == INVALID_INDEX || data.device_y_points[idx] == INVALID_INDEX) {
            data.device_x_points[idx] = INVALID_INDEX;
            data.device_y_points[idx] = INVALID_INDEX;
            return;
        }

        const double lat = data.device_lat_array[idx];
        const double lon = data.device_lon_array[idx];
        double elevation{data.dem_property->no_data_value};
        if (data.dem_type == dem::Type::COPDEM_COG30m) {
            elevation = dem::CopDemCog30mGetElevation(lat, lon, &data.tiles, data.dem_property);
        } else if (data.dem_type == dem::Type::SRTM3) {
            elevation = snapengine::dem::GetElevation(lat, lon, &data.tiles, data.dem_property);
        }

        // TODO: this may need to change if we decide not to use mask.
        if (data.mask_out_area_without_elevation && elevation == data.dem_property->no_data_value) {
            data.device_x_points[idx] = INVALID_INDEX;
            data.device_y_points[idx] = INVALID_INDEX;
        } else {
            (*data.not_null_counter)++;  // race condition is not important here.
        }
    }
}

hipError_t LaunchElevationMask(ElevationMaskData data, hipStream_t stream) {
    dim3 block_size(416);
    dim3 grid_size(cuda::GetGridDim(block_size.x, data.size));

    ElevationMask<<<grid_size, block_size, 0, stream>>>(data);
    return hipGetLastError();
}

}  // namespace backgeocoding
}  // namespace alus
