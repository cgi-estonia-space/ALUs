#include "hip/hip_runtime.h"
/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */
#include "orbit_state_vector.h"
#include "backgeocoding_constants.h"
#include "slave_pixpos.cuh"

#include "earth_gravitational_model96.cuh"
#include "general_constants.h"
#include "geo_utils.cuh"
#include "pos_vector.h"
#include "position_data.h"
#include "sar_geocoding.cuh"
#include "srtm3_elevation_calc.cuh"

/**
 * The contents of this file refer to BackGeocodingOp.computeSlavePixPos in SNAP's java code.
 * They are from s1tbx module.
 */

namespace alus {
namespace backgeocoding{

inline __device__ int GetPosition(s1tbx::DeviceSubswathInfo *subswath_info,
                                  s1tbx::DeviceSentinel1Utils *sentinel1_utils,
                                  int burst_index,
                                  s1tbx::PositionData *position_data,
                                  snapengine::OrbitStateVector *orbit,
                                  const int num_orbit_vec,
                                  const double dt){

    const double zero_doppler_time_in_days = s1tbx::sargeocoding::GetZeroDopplerTime(sentinel1_utils->line_time_interval,
                                                                                 sentinel1_utils->wavelength, position_data->earth_point, orbit, num_orbit_vec, dt);

    if (zero_doppler_time_in_days == s1tbx::sargeocoding::NON_VALID_ZERO_DOPPLER_TIME) {
        return 0;
    }

    const double zero_doppler_time = zero_doppler_time_in_days * snapengine::constants::secondsInDay;
    position_data->azimuth_index =
        burst_index * subswath_info->lines_per_burst +
        (zero_doppler_time - subswath_info->device_burst_first_line_time[burst_index]) /
                                       subswath_info->azimuth_time_interval;

    cudautil::KernelArray<snapengine::OrbitStateVector> orbit_vectors;
    orbit_vectors.array = orbit;
    orbit_vectors.size = num_orbit_vec;
    const double slantRange = s1tbx::sargeocoding::ComputeSlantRangeImpl(
        zero_doppler_time_in_days, orbit_vectors, position_data->earth_point, position_data->sensor_pos);

    if (!sentinel1_utils->srgr_flag) {
        position_data->range_index = (slantRange - subswath_info->slr_time_to_first_pixel * snapengine::constants::lightSpeed) /
            sentinel1_utils->range_spacing;
    } else {
        //TODO: implement this some day, as we don't need it for first demo.
        /*position_data->range_index = s1tbx::sargeocoding::computeRangeIndex(
            su.srgrFlag, su.sourceImageWidth, su.firstLineUTC, su.lastLineUTC,
            su.rangeSpacing, zeroDopplerTimeInDays, slantRange, su.nearEdgeSlantRange, su.srgrConvParams);*/
    }

    if (!sentinel1_utils->near_range_on_left) {
        position_data->range_index = sentinel1_utils->source_image_width - 1 - position_data->range_index;
    }

    return 1;
}

//exclusively supports SRTM3 digital elevation map and none other
__global__ void SlavePixPos(SlavePixPosData calc_data){
    const int idx = threadIdx.x + (blockDim.x * blockIdx.x);
    const int idy = threadIdx.y + (blockDim.y * blockIdx.y);
    double geo_pos_lat;
    double geo_pos_lon;
    double alt;
    s1tbx::PositionData pos_data;

    pos_data.azimuth_index = 0;
    pos_data.range_index = 0;


    if(idx < calc_data.num_pixels && idy < calc_data.num_lines){
        geo_pos_lat = (snapengine::srtm3elevationmodel::RASTER_HEIGHT - calc_data.lat_max_idx + idy) *
                          snapengine::srtm3elevationmodel::DEGREE_RES_BY_NUM_PIXELS_PER_TILE - 60.0;
        geo_pos_lon = (calc_data.lon_min_idx + idx) * snapengine::srtm3elevationmodel::DEGREE_RES_BY_NUM_PIXELS_PER_TILE - 180.0;
        
        alt= snapengine::srtm3elevationmodel::GetElevation(geo_pos_lat, geo_pos_lon, &calc_data.tiles);
        if(alt == calc_data.dem_no_data_value && !calc_data.mask_out_area_without_elevation) {
            alt = snapengine::earthgravitationalmodel96::GetEGM96(
                geo_pos_lat, geo_pos_lon, calc_data.max_lats, calc_data.max_lons, calc_data.egm);
        }

        if(idx==0 && idy == 0){
            printf("altitude number: %f\n", alt); //just getting rid of warnings. Continue working from here.
        }
        if(alt != calc_data.dem_no_data_value ){
            snapengine::geoutils::Geo2xyzWgs84Impl(geo_pos_lat,geo_pos_lon, alt, pos_data.earth_point);

            if(GetPosition(calc_data.device_master_subswath,
                            calc_data.device_master_utils,
                            calc_data.m_burst_index,
                            &pos_data,
                            calc_data.device_master_orbit_state_vectors,
                            calc_data.nr_of_master_vectors,
                            calc_data.master_dt)) {

                calc_data.device_master_az[idy + calc_data.num_lines * idx] = pos_data.azimuth_index;
                calc_data.device_master_rg[idy + calc_data.num_lines * idx] = pos_data.range_index;
                if (GetPosition(calc_data.device_slave_subswath,
                                calc_data.device_slave_utils,
                                calc_data.s_burst_index,
                                &pos_data,
                                calc_data.device_slave_orbit_state_vectors,
                                calc_data.nr_of_slave_vectors,
                                calc_data.slave_dt)) {

                    calc_data.device_slave_az[idy + calc_data.num_lines * idx] = pos_data.azimuth_index;
                    calc_data.device_slave_rg[idy + calc_data.num_lines * idx] = pos_data.range_index;
                    //race condition is not important. we need to know that we have atleast 1 valid index.
                    *calc_data.device_valid_index_counter++;
                }
            }
        }else{
            calc_data.device_master_az[idy + calc_data.num_lines * idx] = INVALID_INDEX;
            calc_data.device_master_rg[idy + calc_data.num_lines * idx] = INVALID_INDEX;
        }
    }
}

hipError_t LaunchSlavePixPos(dim3 grid_size, dim3 block_size, SlavePixPosData calc_data){
    SlavePixPos<<<grid_size, block_size>>>(calc_data);
    return hipGetLastError();
}

} //namespace
} //namespace
