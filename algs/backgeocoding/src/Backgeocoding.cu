#include "Backgeocoding.cuh"
#include "bilinear.cu"
#include "derampDemod.cu"

namespace slap{

hipError_t Backgeocoding::launchBilinear(){
    hipError_t status;
    dim3 gridSize(5,5);
    dim3 blockSize(20,20);

    bilinearInterpolation<<<gridSize, blockSize>>>(
        this->deviceXPoints,
        this->deviceYPoints,
        this->deviceDemodPhase,
        this->deviceDemodI,
        this->deviceDemodQ,
        this->deviceParams,
        0.0,
        this->deviceIResults,
        this->deviceQResults
    );
    status = hipGetLastError();

    return status;
}

//TODO: using placeholder as number 11
hipError_t Backgeocoding::launchDerampDemod(Rectangle slaveRect){
    hipError_t status;
    dim3 gridSize(6,6);
    dim3 blockSize(20,20);

    derampDemod<<<gridSize, blockSize>>>(
        slaveRect,
        this->deviceSlaveI,
        this->deviceSlaveQ,
        this->deviceDemodPhase,
        this->deviceDemodI,
        this->deviceDemodQ,
        this->slaveUtils->subSwath[0],
        11
    );
    status = hipGetLastError();

    return status;
}

}//namespace
