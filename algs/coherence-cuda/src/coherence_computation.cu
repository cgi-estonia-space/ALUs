#include "hip/hip_runtime.h"
/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */

#include <tuple>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <thrust/complex.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "band_params.h"
#include "coh_window.h"
#include "coherence_computation.h"
#include "hip/hip_runtime_api.h"

#include "alus_log.h"

namespace {
struct Power {
    __host__ __device__ double operator()(const double x, const double power) { return pow(x, power); }
};

struct NormalizeDouble {
    static constexpr double hf = 0.5;
    static constexpr double qt = 0.25;
    const double min;
    const double max;
    NormalizeDouble(double min, double max) : min(min), max(max) {}
    __host__ __device__ double operator()(const double& x) { return (x - hf * (min + max)) / (qt * (max - min)); }
};

struct DataMasterNorm {
    __host__ __device__ thrust::tuple<float, float> operator()(const thrust::tuple<float, float, float, float>& t) {
        float master_real = thrust::get<0>(t);
        float master_imaginary = thrust::get<1>(t);
        float slave_real = thrust::get<2>(t);
        float slave_imaginary = thrust::get<3>(t);
        auto out = thrust::complex<float>(master_real, master_imaginary) *
                   thrust::conj(thrust::complex<float>(slave_real, slave_imaginary));
        return thrust::make_tuple(out.real(), out.imag());
    }
};

struct Norm {
    __host__ __device__ thrust::tuple<float> operator()(const thrust::tuple<float, float>& t) {
        float real = thrust::get<0>(t);
        float imaginary = thrust::get<1>(t);
        auto out = (real * real) + (imaginary * imaginary);
        return thrust::make_tuple(out);
    }
};

struct SlaveMultiplyComplexReferencePhase {
    __host__ __device__ thrust::tuple<float, float> operator()(const thrust::tuple<double, float, float>& t) {
        auto flat_earth_phase = thrust::get<0>(t);
        float slave_real = thrust::get<1>(t);
        float slave_imaginary = thrust::get<2>(t);
        auto out = thrust::complex<float>(slave_real, slave_imaginary) *
                   thrust::complex<float>(static_cast<float>(cos(flat_earth_phase)),
                                          static_cast<float>(sin(flat_earth_phase)));
        return thrust::make_tuple(out.real(), out.imag());
    }
};

struct FilteredCoherenceProduct {
    __host__ __device__ thrust::tuple<float> operator()(const thrust::tuple<float, float, float, float, bool>& t) {
        float master_real = thrust::get<0>(t);
        float master_imaginary = thrust::get<1>(t);
        float slave_real = thrust::get<2>(t);
        float slave_imaginary = thrust::get<3>(t);
        bool keep_pixel = thrust::get<4>(t);
        auto product_t = slave_real * slave_imaginary;
        if (keep_pixel && product_t > 0) {
            return thrust::make_tuple(thrust::abs(thrust::complex<float>(master_real, master_imaginary)) /
                                      sqrt(product_t));
        }
        return thrust::make_tuple(0);
    }
};
}  // namespace

namespace alus {
namespace coherence_cuda {

/**
 * input tile contains overlap data but no padding
 * output tile is smaller since overlaps get removed
 */
__global__ void SimpleCoherence2DSumKernelSumSurroundings(float* d_tile_in_data_ptr, float* d_tile_out_data_ptr,
                                                          int input_tile_width, int input_tile_height,
                                                          int output_tile_width, int output_tile_height,
                                                          int coh_window_rg, int coh_window_az, int x_min_pad,
                                                          int x_max_pad, int y_min_pad, int y_max_pad) {
    int column_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    int column_idx_input_tile = column_idx - x_min_pad;
    int row_idx_input_tile = row_idx - y_min_pad;

    if (row_idx < output_tile_height && column_idx < output_tile_width) {
        auto coh_window_rg_size = coh_window_rg;
        auto start_range_x = 0;
        if (column_idx_input_tile < 0) {
            start_range_x = abs(column_idx_input_tile);
        }

        if (column_idx + x_max_pad > input_tile_width - 1) {
            coh_window_rg_size = coh_window_rg + (input_tile_width - 1 - (column_idx + x_max_pad));
        }

        auto coh_window_az_size = coh_window_az;
        auto start_range_y = 0;
        if (row_idx_input_tile < 0) {
            start_range_y = abs(row_idx_input_tile);
        }

        if (row_idx + y_max_pad > input_tile_height - 1) {
            coh_window_az_size = coh_window_az + (input_tile_height - 1 - (row_idx + y_max_pad));
        }

        float pixel_value = 0.0f;
        for (int i = start_range_y; i < coh_window_az_size; i++) {
            float acc = 0.0f;
            for (int j = start_range_x; j < coh_window_rg_size; j++) {
                const int idx = (row_idx_input_tile + i) * input_tile_width + (column_idx_input_tile + j);
                if (idx < (input_tile_width * input_tile_height)) {
                    acc += d_tile_in_data_ptr[idx];
                }
            }
            pixel_value += acc;
        }

        d_tile_out_data_ptr[row_idx * output_tile_width + column_idx] = pixel_value;
    }
}

/**
 * input tile contains overlap data but no padding
 * output tile is smaller since overlaps get removed
 */
__global__ void BoolImageForCoherenceProductFiltering(float* d_tile_in_data_ptr, bool* d_tile_out_data_ptr,
                                                      int input_tile_width, int input_tile_height,
                                                      int output_tile_width, int output_tile_height, int coh_window_rg,
                                                      int coh_window_az, int x_min_pad, int x_max_pad, int y_min_pad,
                                                      int y_max_pad) {
    const float slave_real_no_data = 0.0F;
    auto min_cut_azw = (coh_window_az - 1) / 2;
    auto min_cut_rgw = (coh_window_rg - 1) / 2;

    int column_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    // edge tiles start from same index, other tiles input starts half coherence window before
    int column_idx_input_tile = column_idx + min_cut_rgw - x_min_pad;
    int row_idx_input_tile = row_idx + min_cut_azw - y_min_pad;

    if (row_idx < output_tile_height && column_idx < output_tile_width) {
        auto data_in = d_tile_in_data_ptr[row_idx_input_tile * input_tile_width + column_idx_input_tile];
        d_tile_out_data_ptr[row_idx * output_tile_width + column_idx] =
            std::fabs(data_in - slave_real_no_data) >= std::numeric_limits<float>::epsilon();
    }
}

void CoherenceComputation::Linspace(double min, double max, cuda::DeviceBuffer<double>& d_vector, hipStream_t stream) {
    double delta = (max - min) / static_cast<double>(d_vector.GetElemCount() - 1);
    thrust::transform(thrust::cuda::par.on(stream), thrust::make_counting_iterator(min / delta),
                      thrust::make_counting_iterator((max + 1.) / delta), thrust::make_constant_iterator(delta),
                      d_vector.begin(), thrust::multiplies<double>());
}


void CoherenceComputation::MatMulATransposeB(hipblasHandle_t handle, const double* A, const double* B, double* C,
                                             const int m, const int k, const int n) {
    const double alf = 1;
    const double bet = 0;
    const double* alpha = &alf;
    const double* beta = &bet;
    CHECK_CUDA_ERRORS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha, A, m, B, n, beta, C, m));
}

void CoherenceComputation::LaunchCoherencePreTileCalc(std::vector<int>& x_pows, std::vector<int>& y_pows,
                                                      std::vector<cuda::DeviceBuffer<double>>&& d_burst_coeffs) {
    d_x_pows_ = x_pows;
    d_y_pows_ = y_pows;
    d_ones_ = thrust::device_vector<double>(x_pows.size(), 1.0);
    d_ones_size_ = d_ones_.size();
    subtract_flat_earth_ = true;
    d_burst_coeffs_ = std::move(d_burst_coeffs);
}

void CoherenceComputation::LaunchCoherence(const CohTile& tile, ThreadContext& ctx, const CohWindow& coh_window,
                                           const BandParams& band_params) {
    const int input_tile_width = tile.GetTileIn().GetXSize();
    const int input_tile_height = tile.GetTileIn().GetYSize();
    const int output_tile_width = tile.GetTileOut().GetXSize();
    const int output_tile_height = tile.GetTileOut().GetYSize();

    auto thrust_stream = thrust::cuda::par.on(ctx.stream);
    // how threads are in block
    //    dim3 threads_per_block(32, 32, 1);  // this should be 32 <= x*y*z <=1024 & warp size based (multiple of 32)
    //    int output_size = output_tile_width * output_tile_height;

    // thread block limits: x_max:1024, y_max:1024 and z_max:64 && x × y × z ≤ 1024
    // x_width_max * y_limit_max = 1024
    dim3 threads_per_block(32, 32, 1);
    dim3 num_blocks((output_tile_width + threads_per_block.x - 1) / threads_per_block.x,
                    (output_tile_height + threads_per_block.y - 1) / threads_per_block.y);

    ctx.d_tile_out_slave_real_bool.Resize(output_tile_width * output_tile_height);


    BoolImageForCoherenceProductFiltering<<<num_blocks, threads_per_block, 0, ctx.stream>>>(
        ctx.d_band_slave_real.Get(), ctx.d_tile_out_slave_real_bool.Get(), input_tile_width, input_tile_height,
        output_tile_width, output_tile_height, coh_window.rg, coh_window.az, tile.GetXMinPad(), tile.GetXMaxPad(),
        tile.GetYMinPad(), tile.GetYMaxPad());
    CHECK_CUDA_ERRORS(hipPeekAtLastError());

    size_t size_to_last = input_tile_height * input_tile_width;
    // ComputeFlatEarthPhase
    if (subtract_flat_earth_) {
        ctx.d_range_axis.Resize(input_tile_width);
        ctx.d_azimuth_axis.Resize(input_tile_height);
        const auto& tile_in = tile.GetTileIn();
        Linspace(tile_in.GetXMin(), tile_in.GetXMax(), ctx.d_range_axis, ctx.stream);
        Linspace(0, tile.GetBurstSize() - 1, ctx.d_azimuth_axis, ctx.stream);


        thrust::transform(thrust_stream, ctx.d_range_axis.begin(), ctx.d_range_axis.end(), ctx.d_range_axis.begin(),
                          NormalizeDouble(band_params.band_x_min, band_params.band_x_size - 1));
        thrust::transform(thrust_stream, ctx.d_azimuth_axis.begin(), ctx.d_azimuth_axis.end(),
                          ctx.d_azimuth_axis.begin(),
                          NormalizeDouble(band_params.band_y_min, /*band_params.band_y_size - 1 */ tile.GetBurstSize() - 1));

        hipblasHandle_t handle = ctx.handle;

        ctx.d_x_xpows_ones_t.Resize(d_ones_.size() * ctx.d_range_axis.size());
        ctx.d_y_ypows_ones_t.Resize(d_ones_.size() * ctx.d_azimuth_axis.size());

        size_t d_azimuth_axis_size = ctx.d_azimuth_axis.size();
        size_t d_range_axis_size = ctx.d_range_axis.size();
        // 2675*21
        MatMulATransposeB(handle, ctx.d_range_axis.data(), thrust::raw_pointer_cast(d_ones_.data()),
                          ctx.d_x_xpows_ones_t.data(), d_range_axis_size, 1, d_ones_.size());
        // 1503*21
        MatMulATransposeB(handle, ctx.d_azimuth_axis.data(), thrust::raw_pointer_cast(d_ones_.data()),
                          ctx.d_y_ypows_ones_t.data(), d_azimuth_axis_size, 1, d_ones_.size());

        ctx.d_y_ones_ypows_t.Resize(d_ones_size_ * d_azimuth_axis_size);
        ctx.d_x_ones_xpows_t.Resize(d_ones_size_ * d_range_axis_size);

        // SWAPED POWS HERE ON PURPOSE
        ctx.d_y_ones.Resize(d_azimuth_axis_size);
        ctx.d_x_ones.Resize(d_range_axis_size);
        thrust::fill(thrust_stream, ctx.d_y_ones.begin(), ctx.d_y_ones.end(), 1.0);
        thrust::fill(thrust_stream, ctx.d_x_ones.begin(), ctx.d_x_ones.end(), 1.0);

        // 2675*21
        MatMulATransposeB(handle, ctx.d_x_ones.data(), thrust::raw_pointer_cast(d_y_pows_.data()),
                          ctx.d_x_ones_xpows_t.data(), ctx.d_x_ones.size(), 1, d_y_pows_.size());

        // 1503*21
        MatMulATransposeB(handle, ctx.d_y_ones.data(), thrust::raw_pointer_cast(d_x_pows_.data()),
                          ctx.d_y_ones_ypows_t.data(), ctx.d_y_ones.size(), 1, d_x_pows_.size());

        // POWER X AND Y SIDES
        thrust::transform(thrust_stream, ctx.d_x_xpows_ones_t.begin(), ctx.d_x_xpows_ones_t.end(),
                          ctx.d_x_ones_xpows_t.begin(), ctx.d_x_xpows_ones_t.begin(), Power());
        thrust::transform(thrust_stream, ctx.d_y_ypows_ones_t.begin(), ctx.d_y_ypows_ones_t.end(),
                          ctx.d_y_ones_ypows_t.begin(), ctx.d_y_ypows_ones_t.begin(), Power());

        auto& d_coefs = d_burst_coeffs_.at(tile.GetBurstIndex());
        ctx.d_y_ones_coefs_t.Resize(d_coefs.size() * ctx.d_y_ones.size());

        // Y SIDE COEFS MULTIPLY Y_SIDE_POWERS
        // make coefs same shape
        MatMulATransposeB(handle, ctx.d_y_ones.data(), d_coefs.data(), ctx.d_y_ones_coefs_t.data(), ctx.d_y_ones.size(),
                          1, d_coefs.size());

        // multiply y side by coefs
        thrust::transform(thrust_stream, ctx.d_y_ypows_ones_t.begin(), ctx.d_y_ypows_ones_t.end(),
                          ctx.d_y_ones_coefs_t.begin(), ctx.d_y_ypows_ones_t.begin(), thrust::multiplies<double>());

        ctx.d_flat_earth_phase.Resize(d_range_axis_size * d_azimuth_axis_size);

        MatMulATransposeB(handle, ctx.d_x_xpows_ones_t.data(), ctx.d_y_ypows_ones_t.data(),
                          ctx.d_flat_earth_phase.data(), d_range_axis_size, d_ones_size_, d_azimuth_axis_size);

        thrust::transform(
            thrust_stream,
            thrust::make_zip_iterator(thrust::make_tuple(ctx.d_flat_earth_phase.begin(), ctx.d_band_slave_real.begin(),
                                                         ctx.d_band_slave_imag.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(ctx.d_flat_earth_phase.end(),
                                                         ctx.d_band_slave_real.end() /*+ size_to_last*/,
                                                         ctx.d_band_slave_imag.end())),
            thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_slave_real.begin(), ctx.d_band_slave_imag.begin())),
            SlaveMultiplyComplexReferencePhase());
    }

    ctx.complex_data_slave_norm_imaginary.Resize(input_tile_height * input_tile_width);

    thrust::transform(
        thrust_stream,
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_master_real.begin(), ctx.d_band_master_imag.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_master_real.end(), ctx.d_band_master_imag.end())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.complex_data_slave_norm_imaginary.begin())), Norm());

    ctx.complex_data_slave_norm_real.Resize(input_tile_height * input_tile_width);

    thrust::transform(
        thrust_stream,
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_slave_real.begin(), ctx.d_band_slave_imag.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_slave_real.end(), ctx.d_band_slave_imag.end())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.complex_data_slave_norm_real.begin())), Norm());

    // input is master_real,master_imaginary,slave_real,slave_imaginary
    // output is data_master_norm (check coherence_calc)
    ctx.data_master_norm_real.Resize(size_to_last);
    ctx.data_master_norm_imaginary.Resize(size_to_last);

    thrust::transform(
        thrust_stream,
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_master_real.begin(), ctx.d_band_master_imag.begin(),
                                                     ctx.d_band_slave_real.begin(), ctx.d_band_slave_imag.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_band_master_real.end(), ctx.d_band_master_imag.end(),
                                                     ctx.d_band_slave_real.end(), ctx.d_band_slave_imag.end())),
        thrust::make_zip_iterator(
            thrust::make_tuple(ctx.data_master_norm_real.begin(), ctx.data_master_norm_imaginary.begin())),
        DataMasterNorm());

    // todo: use streams to make custom kernel per layer device level parallel

    ctx.d_tile_out_master_real.Resize(output_tile_width * output_tile_height);
    ctx.d_tile_out_master_imag.Resize(output_tile_width * output_tile_height);
    ctx.d_tile_out_slave_real.Resize(output_tile_width * output_tile_height);
    ctx.d_tile_out_slave_imag.Resize(output_tile_width * output_tile_height);

    // 2d kernel sum block dimensions improve performance, seems longer x dim is always better
    // even in the unlikely scenario where azimuth window is larger - most likely due to memory access patters,
    // could use shared memory in the future as an optimization
    dim3 sum_block_dim(32, 4);
    dim3 sum_num_blocks((output_tile_width + sum_block_dim.x - 1) / sum_block_dim.x,
                        (output_tile_height + sum_block_dim.y - 1) / sum_block_dim.y);

    SimpleCoherence2DSumKernelSumSurroundings<<<sum_num_blocks, sum_block_dim, 0, ctx.stream>>>(
        ctx.data_master_norm_real.data(), ctx.d_tile_out_master_real.data(), input_tile_width, input_tile_height,
        output_tile_width, output_tile_height, coh_window.rg, coh_window.az, tile.GetXMinPad(), tile.GetXMaxPad(),
        tile.GetYMinPad(), tile.GetYMaxPad());
    CHECK_CUDA_ERRORS(hipPeekAtLastError());

    SimpleCoherence2DSumKernelSumSurroundings<<<sum_num_blocks, sum_block_dim, 0, ctx.stream>>>(
        ctx.data_master_norm_imaginary.data(), ctx.d_tile_out_master_imag.data(), input_tile_width, input_tile_height,
        output_tile_width, output_tile_height, coh_window.rg, coh_window.az, tile.GetXMinPad(), tile.GetXMaxPad(),
        tile.GetYMinPad(), tile.GetYMaxPad());
    CHECK_CUDA_ERRORS(hipPeekAtLastError());

    SimpleCoherence2DSumKernelSumSurroundings<<<sum_num_blocks, sum_block_dim, 0, ctx.stream>>>(
        ctx.complex_data_slave_norm_real.data(), ctx.d_tile_out_slave_real.data(), input_tile_width, input_tile_height,
        output_tile_width, output_tile_height, coh_window.rg, coh_window.az, tile.GetXMinPad(), tile.GetXMaxPad(),
        tile.GetYMinPad(), tile.GetYMaxPad());
    CHECK_CUDA_ERRORS(hipPeekAtLastError());

    SimpleCoherence2DSumKernelSumSurroundings<<<sum_num_blocks, sum_block_dim, 0, ctx.stream>>>(
        ctx.complex_data_slave_norm_imaginary.data(), ctx.d_tile_out_slave_imag.data(), input_tile_width,
        input_tile_height, output_tile_width, output_tile_height, coh_window.rg, coh_window.az, tile.GetXMinPad(),
        tile.GetXMaxPad(), tile.GetYMinPad(), tile.GetYMaxPad());
    CHECK_CUDA_ERRORS(hipPeekAtLastError());

    ctx.d_tile_out.Resize(output_tile_height * output_tile_width);

    thrust::transform(
        thrust_stream,
        thrust::make_zip_iterator(thrust::make_tuple(
            ctx.d_tile_out_master_real.begin(), ctx.d_tile_out_master_imag.begin(), ctx.d_tile_out_slave_real.begin(),
            ctx.d_tile_out_slave_imag.begin(), ctx.d_tile_out_slave_real_bool.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_tile_out_master_real.end(), ctx.d_tile_out_master_imag.end(),
                                                     ctx.d_tile_out_slave_real.end(), ctx.d_tile_out_slave_imag.end(),
                                                     ctx.d_tile_out_slave_real_bool.end())),
        thrust::make_zip_iterator(thrust::make_tuple(ctx.d_tile_out.begin())), FilteredCoherenceProduct());
}

}  // namespace coherence_cuda
}  // namespace alus