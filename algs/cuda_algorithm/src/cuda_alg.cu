#include "hip/hip_runtime.h"
#include <algorithm>
#include <array>
#include <iostream>
#include <stdio.h>

#include <cuda/hip/hip_runtime.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/hip/hip_runtime_api.h>


__global__ void square(float* dOut, float* dIn)
{
    int threadIndex = threadIdx.x;
    float f = dIn[threadIndex];
    dOut[threadIndex] = f * f;
}

__global__ void cube(float* dOut, float* dIn)
{
    int threadIndex = threadIdx.x;
    float f = dIn[threadIndex];
    dOut[threadIndex] = f * f * f;
}

void cuda_kernel()
{
    constexpr size_t SERIES_SPAN{200};
    constexpr size_t SERIES_BUFFER_SIZE{SERIES_SPAN * sizeof(float)};

    // Host data buffers.
    std::array<float, SERIES_SPAN> hIn;
    std::iota(hIn.begin(), hIn.end(), 0);
    std::array<float, SERIES_SPAN> hOut;

    float* dIn;
    float* dOut;

    hipMalloc((void**)&dIn, SERIES_BUFFER_SIZE);
    hipMalloc((void**)&dOut, SERIES_BUFFER_SIZE);

    hipMemcpy(dIn, hIn.data(), SERIES_BUFFER_SIZE, hipMemcpyHostToDevice);

    cube<<<1, SERIES_SPAN>>>(dOut, dIn);

    hipMemcpy(hOut.data(), dOut, SERIES_BUFFER_SIZE, hipMemcpyDeviceToHost);

    int col{0};
    for (auto const result : hOut)
    {
        std::cout << result;
        if (col % 4 != 3)
            std::cout << '\t' << '\t' << '\t';
        else
            std::cout << std::endl;
        col++;
    }

    hipFree(dIn);
    hipFree(dOut);
}